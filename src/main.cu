#include "common.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <functional>

double time_once(std::function<void()> func) {
  static Timer timer;
  CUDA_CALL(hipDeviceSynchronize());
  timer.start();
  func();
  CUDA_CALL(hipDeviceSynchronize());
  timer.end();
  return timer.time_in_second();
}

i32 main() { return 0; }
