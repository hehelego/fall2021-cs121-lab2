#include "common.cuh"
#include "cuckoo_cpu.cuh"
#include "cuckoo_gpu.cuh"

#include <functional>
#include <iomanip>
#include <set>
#include <vector>

double time_once(std::function<void()> func) {
  static Timer timer;
  CUDA_CALL(hipDeviceSynchronize());
  timer.start();
  func();
  CUDA_CALL(hipDeviceSynchronize());
  timer.end();
  return timer.time_in_second();
}
double time_func(std::function<void()> func, u32 runs) {
  double s = 0;
  for (u32 i = 0; i < runs; i++) s += time_once(func);
  return s / runs;
}

int main(){ return 0; }