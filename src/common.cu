#include "common.cuh"
#include <random>

u32 getRandSeed() {
  std::random_device rand_dev;
  return rand_dev();
}

void batchRandomGen(u32 *d_array, u32 n) {
  hiprandGenerator_t rng;
  CURAND_CALL(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_MT19937));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(rng, getRandSeed()));
  CURAND_CALL(hiprandGenerate(rng, d_array, n));
}
