#include "hip/hip_runtime.h"
#include "common.cuh"
#include "cuckoo_gpu.cuh"

#include <utility>

namespace GpuTable {
// test whether x is 0xFFFFFFFF
static __device__ inline bool empty(u32 x) { return (~x) == 0u; }
static const u32 EMPTY = (~0u);

static __global__ void rollBackKernel(std::pair<u32, u32> *result, u32 n, u32 **slots) {
  u32 i = threadIdx.x + blockIdx.x * blockDim.x;
  u32 final_table = EMPTY, final_slot = EMPTY;
  if (i < n) {
    final_table = result[i].first;
    final_slot = result[i].second;
    if (!empty(final_table) && !empty(final_slot)) slots[final_table][final_slot] = EMPTY;
  }
}
static __global__ void updateKernel(u32 *keys, std::pair<u32, u32> *result, u32 n, u32 cap, u32 **slots, const u32 *const seeds,
                                    u32 m, u32 threshold, u32 *counter) {
  u32 cachedSeeds[M_HASH_FUNCS];
  for (u32 i = 0; i < M_HASH_FUNCS; i++) cachedSeeds[i] = seeds[i];

  u32 i = threadIdx.x + blockIdx.x * blockDim.x;
  u32 key = 0, final_table = EMPTY, final_slot = EMPTY;
  if (i < n) {
    key = keys[i];
    for (u32 j = 0; !empty(key) && j < threshold; j++) {
      u32 jj = j % m;
      u32 slot = xxHash32(cachedSeeds[jj], key) % cap;
      final_table = jj, final_slot = slot;
      key = atomicExch(&slots[jj][slot], key);
    }
    if (!empty(key)) {
      atomicAdd(counter, 1);
      result[i].first = EMPTY;
      result[i].second = EMPTY;
    } else {
      result[i].first = final_table;
      result[i].second = final_slot;
    }
  }
}
static __global__ void queryKernel(const u32 *keys, u32 *result, u32 n, u32 cap, u32 **const slots, const u32 *const seeds,
                                   u32 m) {
  u32 cachedSeeds[M_HASH_FUNCS];
  for (u32 i = 0; i < M_HASH_FUNCS; i++) cachedSeeds[i] = seeds[i];

  u32 i = threadIdx.x + blockIdx.x * blockDim.x;
  u32 key = 0;
  if (i < n && !empty(keys[i])) {
    key = keys[i];
    for (u32 j = 0; j < m; j++) {
      u32 slot = xxHash32(cachedSeeds[j], key) % cap;
      result[i] += slots[j][slot] == key;
    }
  }
}
static inline u32 queryMaxThreadsPerBlock() {
  hipDeviceProp_t dp;
  CUDA_CALL(hipGetDeviceProperties(&dp, 0));
  return dp.maxThreadsPerBlock;
}
Table::Table(u32 cap, u32 t) : THREADS_PER_BLOCK(queryMaxThreadsPerBlock()) {
  _n = cap, _m = t;
  _threshold = binaryLength(_n) * 4;
  _slots = coda::malloc<u32 *>(_m);
  for (u32 i = 0; i < _m; i++) _slotsHost[i] = coda::malloc<u32>(_n);
  coda::copy(_slots, _slotsHost, _m, coda::H2D);
  _seeds = coda::malloc<u32>(M_HASH_FUNCS);
  clear();
}

Table::~Table() {
  coda::copy(_slotsHost, _slots, _m, coda::D2H);
  for (u32 i = 0; i < _m; i++) coda::free(_slotsHost[i]);
  coda::free(_slots);
  coda::free(_seeds);
}

void Table::clear() {
  coda::randomArray(_seeds, _m);
  _sz = 0;
  coda::copy(_slotsHost, _slots, _m, coda::D2H);
  for (u32 i = 0; i < _m; i++) coda::fill0xFF(_slotsHost[i], _n);
}

void Table::rehash() {
  Debug() << "GPU TABLE: rehash\n";
  u32 *backup[M_HASH_FUNCS];
  coda::copy(_slotsHost, _slots, _m, coda::D2H);
  for (u32 i = 0; i < _m; i++) {
    backup[i] = _slotsHost[i];
    _slotsHost[i] = coda::malloc<u32>(_n);
  }
  coda::copy(_slots, _slotsHost, _m, coda::H2D);
  clear();
  for (u32 i = 0; i < _m; i++) update(backup[i], _n);
  for (u32 i = 0; i < _m; i++) coda::free(backup[i]);
}

void Table::update(u32 *keys, u32 n) {
  u32 blocks = div_ceil(n, THREADS_PER_BLOCK);
  u32 *d_counter = coda::malloc<u32>(1), counter = 0;
  auto result = coda::malloc<std::pair<u32, u32>>(n);
  while (true) {
    coda::fillZero(d_counter, 1);
    updateKernel<<<blocks, THREADS_PER_BLOCK>>>(keys, result, n, _n, _slots, _seeds, _m, _threshold, d_counter);
    coda::copy(&counter, d_counter, 1, coda::D2H);
    if (counter > 0) {
      rollBackKernel<<<blocks, THREADS_PER_BLOCK>>>(result, n, _slots);
      rehash();
    } else {
      break;
    }
  }
  coda::free(d_counter), coda::free(result);

  _sz += n;
}
void Table::query(u32 *keys, u32 *result, u32 n) const {
  u32 blocks = div_ceil(n, THREADS_PER_BLOCK);
  coda::fillZero(result, n);
  queryKernel<<<blocks, THREADS_PER_BLOCK>>>(keys, result, n, _n, _slots, _seeds, _m);
}

} // namespace GpuTable
