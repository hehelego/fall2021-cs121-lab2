#include "common.cuh"
#include "cuckoo_cpu.cuh"
#include "cuckoo_gpu.cuh"

#include <functional>
#include <iomanip>
#include <set>
#include <vector>

double time_once(std::function<void()> func) {
  static Timer timer;
  CUDA_CALL(hipDeviceSynchronize());
  timer.start();
  func();
  CUDA_CALL(hipDeviceSynchronize());
  timer.end();
  return timer.time_in_second();
}
double time_func(std::function<void()> func, u32 runs) {
  double s = 0;
  for (u32 i = 0; i < runs; i++) s += time_once(func);
  return s / runs;
}

void checkCpuTable() {
  const u32 N = 1 << 20;
  const u32 M = 1 << 13;
  CpuTable::Table t_cpu(N * 2, 2);
  CpuTable::UnorderedMap t_stl;

  u32 *key = new u32[N], *qry = new u32[N];
  u32 *res0 = new u32[N], *res1 = new u32[N];

  // auto f = [&]() {
    randomArray(key, N), randomArray(qry, N);

    u32 *sqry = new u32[M];
    randomArray(sqry, M);
    for (u32 i = 0; i < M; i++) qry[sqry[i] % N] = key[sqry[i] % N];
    delete[] sqry;

    t_cpu.clear(), t_stl.clear();
    t_cpu.update(key, N), t_stl.update(key, N);
    t_cpu.query(qry, res0, N), t_stl.query(qry, res1, N);
    bool cmp = std::equal(res0, res0 + N, res1, res1 + N);
    if(!cmp) std::abort();
  // };
  // time_func(f, 10);

  delete[] key, delete[] qry;
  delete[] res0, delete[] res1;
  return;
}
void checkGpuTable() {
  const u32 N = 1 << 20;
  const u32 M = 1 << 13;
  GpuTable::Table t_gpu(N * 2, 2);
  CpuTable::UnorderedMap t_stl;

  u32 *hostKey = new u32[N], *hostQry = new u32[N];
  u32 *deviceKey = coda::malloc<u32>(N), *deviceQry = coda::malloc<u32>(N);
  u32 *hostResult = new u32[N], *hostGpuResult = new u32[N];
  u32 *deviceResult = coda::malloc<u32>(N);

  // auto f = [&]() {
    coda::randomArray(deviceKey, N), coda::randomArray(deviceQry, N), coda::copy(deviceQry, deviceKey, M, coda::D2D);
    coda::copy(hostKey, deviceKey, N, coda::D2H), coda::copy(hostQry, deviceQry, N, coda::D2H);

    t_gpu.clear(), t_stl.clear();
    t_gpu.update(deviceKey, N), t_stl.update(hostKey, N);
    t_gpu.query(deviceQry, deviceResult, N), t_stl.query(hostQry, hostResult, N);
    coda::copy(hostGpuResult, deviceResult, N, coda::D2H);

    bool cmp = std::equal(hostGpuResult, hostGpuResult + N, hostResult, hostResult + N);
    if(!cmp) std::abort();
  // };
  // time_func(f, 10);

  delete[] hostKey, delete[] hostQry, delete[] hostResult, delete[] hostGpuResult;
  coda::free(deviceKey), coda::free(deviceQry), coda::free(deviceResult);
  return;
}

void checkBatchUpdate() {
  // for k>=21, likely to trigger infinity rehash.
  u32 k = 0;
  std::cin >> k;
  const u32 N = 1 << k;
  GpuTable::Table t_gpu(N * 2, 2);

  u32 *deviceKey = coda::malloc<u32>(N);
  u32 *hostKey = new u32[N];
  coda::randomArray(deviceKey, N);
  coda::copy(hostKey, deviceKey, N, coda::D2H);

  // freopen("input","w",stdout);
  // for(u32 i=0;i<N;i++) printf("%u ", hostKey[i]);
  // fflush(stdout);


  CUDA_CALL(hipDeviceSynchronize());

  Debug() << "GO\n";
  t_gpu.update(deviceKey, N);
  Debug() << "END\n";

  CUDA_CALL(hipDeviceSynchronize());

  coda::free(deviceKey);
  delete[] hostKey;
  return;
}

int main(){
  checkCpuTable();
  checkGpuTable();
  return 0;
}
